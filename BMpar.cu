#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <string>
#include <cmath>
#include <ctime>
#include <ctime>
#include <cstring>
#include <stdlib.h>
#include <stdio.h>
#include <random>
#include <chrono>

#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

using namespace std;

#define N 256

__global__ void BMwp(float f_new[], float dw[], const float dt, hiprandState *state)
{
  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  dw[tid] = dt*getrand(&state[tid]);
  f_new[tid+1] = f_new[tid] + dw[tid];
}

__global__ void initialize(float f[], float x[])
{
    int tid = threadIdx.x + blockIdx.x*blockDim.x;

	  x[tid] = 0.0f;
	  f[tid] = 0.0f;
}

__global__ void init_r(hiprandState *state, unsigned long seed)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  hiprand_init(seed,0,0, &state[idx]);
}
__device__ float getrand(hiprandState *state)
{
  return (float)(hiprand_normal(state));
}

void io_fun(std::string file, float *x, const int N)
{
 std::ofstream myfile_tsN;
 myfile_tsN.open(file);
 for(int i = 0; i< N; i++)
 {
   myfile_tsN << x[i] << std::endl;
 }
 myfile_tsN.close();
}

int main()
{
  const int T = 1;
  const float dt = sqrt(T/float(N));

  size_t sz = N*sizeof(float);

  float *f, *dw, *devstate;
  f  = new float[N];
  dw = new float[N];
  rin= new float[N];


  float *d_f, *d_dw;
  hipMalloc(&d_f,sz);
  hipMalloc(&d_dw,sz);
  hipMalloc(&devstate, sz)

  dim3 dimBlock(16,1,1);
  dim3 dimGrid(N/dimBlock.x,1,1);

  initialize<<<dimGrid, dimBlock>>>(d_f, d_dw);
  cudaDeviceSynchonize();
  init_r<<<dimGrid, dimBlock>>>(devstate,0);
  cudaDeviceSynchonize();

  std::chrono::steady_clock::time_point start = std::chrono::steady_clock::now();
  BMwp<<<dimGrid, dimBlock>>>(d_f, d_dw, dt, devstate);
  cudaDeviceSynchonize();
  std::chrono::steady_clock::time_point end = chrono::steady_clock::now();

  cout << "Parallel Weiner Process time in microseconds: "
       << chrono::duration_cast<chrono::microseconds>(end - start).count()
       << " microseconds" << endl;

  hipMemcpy(f, d_f,sz, hipMemcpyDeviceToHost);
  hipMemcpy(dw, d_dw,sz,hipMemcpyDeviceToHost);


  std::string f3;
  f3 = "PARwp.dat";

  io_fun(f3, f, N);

  delete f, dw;

  hipFree(d_f);
  hipFree(devstate);
  hipFree(d_dw);
}
